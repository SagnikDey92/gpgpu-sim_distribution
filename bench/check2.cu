
#include <hip/hip_runtime.h>
#include <stdio.h>

#define NBLOCKS  2
#define TPERBLK  1

#define NTHREADS (NBLOCKS * TPERBLK)

void errCheck()
{
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error %d: %s\n", err, hipGetErrorString(err));
        exit(1);
    }
}


// @@@code  {inter block atomic race}   {cuda:bbatomic}
__device__ int lock = 0;
__device__ int dummy = 0;

__global__ void kmain(unsigned int *data)   // @@@{
{
while (atomicCAS(&lock, 0, 1) != 0)
__threadfence();
data[1]++;
__threadfence();
atomicExch(&lock, 0);
data[0]++;
}                                           // @@@}

int main() 
{
    int N = 5;
    unsigned int *d_data;
    hipMalloc(&d_data, N*sizeof(unsigned int));
    unsigned int *t = (unsigned int*)malloc(N*sizeof(unsigned int));
    for (int i = 0; i < N; ++i)
       t[i] = 0;
    hipMemcpy(d_data, t, N*sizeof(unsigned int), hipMemcpyHostToDevice);
    kmain<<<NBLOCKS,TPERBLK>>>(d_data);
    hipMemcpy(t, d_data, N*sizeof(unsigned int), hipMemcpyDeviceToHost);
    printf("%u, %u\n", t[0], t[1]);
    errCheck();
    return 0;
}
