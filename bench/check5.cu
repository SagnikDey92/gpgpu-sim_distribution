
#include <hip/hip_runtime.h>
#include <stdio.h>

#define NBLOCKS  5
#define TPERBLK  1

#define NTHREADS (NBLOCKS * TPERBLK)

void errCheck()
{
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error %d: %s\n", err, hipGetErrorString(err));
        exit(1);
    }
}


// @@@code  {inter block atomic race}   {cuda:bbatomic}
__device__ int flag = 0;
__device__ int dummy = 0;

__global__ void kmain(unsigned int *data)   // @@@{
{
    data[0]++;
}                                           // @@@}

int main() 
{
    unsigned int *d_data;
    hipMalloc(&d_data, sizeof(unsigned int));
    unsigned int *t = (unsigned int*)malloc(sizeof(unsigned int));
    t[0] = 0;
    hipMemcpy(d_data, t, sizeof(unsigned int), hipMemcpyHostToDevice);
    kmain<<<NBLOCKS,TPERBLK>>>(d_data);
    hipMemcpy(t, d_data, sizeof(int), hipMemcpyDeviceToHost);
    printf("%u\n", t[0]);
    errCheck();
    return 0;
}
